#include "error.cuh"
#include <stdio.h>
#include <stdint.h>

const int N = 30;

int main(void)
{
    for (int n = 1; n <= N; ++n)
    {
        const size_t size = size_t(n) * 1024 * 1024 * 1024;
        uint64_t *x;
#ifdef UNIFIED    //统一内存超量分配
        CHECK(hipMallocManaged(&x, size));
        CHECK(hipFree(x));
        printf("Allocated %d GB unified memory without touch.\n", n);
#else //只分配GPU内存
        CHECK(hipMalloc(&x, size));
        CHECK(hipFree(x));
        printf("Allocate %d GB device memory.\n", n);
#endif
    }
    return 0;
}


