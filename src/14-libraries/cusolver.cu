#include "error.cuh" 
#include <stdio.h>
#include <stdlib.h>
#include <hipsolver.h>

int main(void)
{
    int N = 2;
    int N2 = N * N;

    hipDoubleComplex *A_cpu = (hipDoubleComplex *) 
        malloc(sizeof(hipDoubleComplex) * N2);    // 分配复数矩阵内存
    for (int n = 0; n < N2; ++n)  //对复数矩阵赋值
    {
        A_cpu[0].x = 0;
        A_cpu[1].x = 0;
        A_cpu[2].x = 0;
        A_cpu[3].x = 0;
        A_cpu[0].y = 0; 
        A_cpu[1].y = 1;
        A_cpu[2].y = -1;
        A_cpu[3].y = 0;
    }
    hipDoubleComplex *A;  // 定义device的矩阵
    CHECK(hipMalloc((void**)&A, sizeof(hipDoubleComplex) * N2)); // 分配显存的
    CHECK(hipMemcpy(A, A_cpu, sizeof(hipDoubleComplex) * N2,    // 数据传输的，host 到 device
        hipMemcpyHostToDevice));

    double *W_cpu = (double*) malloc(sizeof(double) * N);        // 分配本征值的内存
    double *W; 
    CHECK(hipMalloc((void**)&W, sizeof(double) * N));     //  分配本征值的显存

    hipsolverHandle_t handle = NULL;  
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    int lwork = 0;
// 确定运算需要多少缓冲空间
    hipsolverDnZheevd_bufferSize(handle, jobz, uplo, 
        N, A, N, W, &lwork);
    hipDoubleComplex* work;
    CHECK(hipMalloc((void**)&work, 
        sizeof(hipDoubleComplex) * lwork));   // 分配缓冲空间显存

    int* info;
    CHECK(hipMalloc((void**)&info, sizeof(int)));     //  返回值
    hipsolverDnZheevd(handle, jobz, uplo, N, A, N, W, 
        work, lwork, info);                                // 算出本征值的
    hipMemcpy(W_cpu, W, sizeof(double) * N, 
        hipMemcpyDeviceToHost);

    printf("Eigenvalues are:\n");
    for (int n = 0; n < N; ++n)
    {
        printf("%g\n", W_cpu[n]);
    }

    hipsolverDnDestroy(handle);

    free(A_cpu);
    free(W_cpu);
    CHECK(hipFree(A));
    CHECK(hipFree(W));
    CHECK(hipFree(work));
    CHECK(hipFree(info));

    return 0;
}
