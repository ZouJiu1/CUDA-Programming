#include <thrust/execution_policy.h>
#include <thrust/scan.h>
#include <stdio.h>

int main(void)
{
    int N = 10;
    int *x, *y;
    hipMalloc((void **)&x, sizeof(int) * N);
    hipMalloc((void **)&y, sizeof(int) * N);
    int *h_x = (int*) malloc(sizeof(int) * N);
    for (int i = 0; i < N; ++i)
    {
        h_x[i] = i + 1;
    }
    hipMemcpy(x, h_x, sizeof(int) * N, hipMemcpyHostToDevice); // 复制到device

    thrust::inclusive_scan(thrust::device, x, x + N, y);      //使用了指针的包含扫描，重载函数的

    int *h_y = (int*) malloc(sizeof(int) * N);
    hipMemcpy(h_y, y, sizeof(int) * N, hipMemcpyDeviceToHost);  //复制到host
    for (int i = 0; i < N; ++i)
    {
        printf("%d ", h_y[i]);
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    free(h_x);
    free(h_y);
    return 0;
}

